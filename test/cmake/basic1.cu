// adapted from cudnn-training  https://github.com/tbennun/cudnn-training/

#include <iostream>

#include <hip/hip_runtime.h>


#include <hipblas.h>
#include <cudnn.h>

void checkCudaErrors(size_t status) {
    if(status != 0) {
        std::cout << "ERROR status non-zero: " << status << std::endl;
    }
}

void checkCUDNN(size_t status) {
    if(status != 0) {
        std::cout << "ERROR status non-zero: " << status << std::endl;
    }
}

int main(int argc, char *argv[]) {
    cudnnHandle_t cudnnHandle;
    hipblasHandle_t cublasHandle;
    cudnnTensorDescriptor_t dataTensor;

    // Create CUBLAS and CUDNN handles
    int gpuid = 0;
    checkCudaErrors(hipSetDevice(gpuid));
    checkCudaErrors(hipblasCreate(&cublasHandle));
    checkCUDNN(cudnnCreate(&cudnnHandle));

    // Create tensor descriptors
    checkCUDNN(cudnnCreateTensorDescriptor(&dataTensor));

    return 0;
}
